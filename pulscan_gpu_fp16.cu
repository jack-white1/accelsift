#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hipfft/hipfft.h>
#include <hip/hip_fp16.h>

extern "C" {
#include "localcdflib.h"
}

struct candidate{
    float power;
    float logp;
    int r;
    int z;
    int numharm;
};

typedef struct {
    half power;
    uint16_t index;
} power_index_struct;


// comparison function for qsort
int compareCandidatesByLogp(const void* a, const void* b){
    candidate* candidateA = (candidate*)a;
    candidate* candidateB = (candidate*)b;
    if (candidateA->logp > candidateB->logp){
        return 1;
    } else if (candidateA->logp < candidateB->logp){
        return -1;
    } else {
        return 0;
    }
}

double extended_equiv_gaussian_sigma(double logp)
/*
  extended_equiv_gaussian_sigma(double logp):
      Return the equivalent gaussian sigma corresponding to the 
          natural log of the cumulative gaussian probability logp.
          In other words, return x, such that Q(x) = p, where Q(x)
          is the cumulative normal distribution.  This version uses
          the rational approximation from Abramowitz and Stegun,
          eqn 26.2.23.  Using the log(P) as input gives a much
          extended range.
*/
{
    double t, num, denom;

    t = sqrt(-2.0 * logp);
    num = 2.515517 + t * (0.802853 + t * 0.010328);
    denom = 1.0 + t * (1.432788 + t * (0.189269 + t * 0.001308));
    return t - num / denom;
}

double equivalent_gaussian_sigma(double logp)
/* Return the approximate significance in Gaussian sigmas */
/* corresponding to a natural log probability logp        */
{
    double x;

    if (logp < -600.0) {
        x = extended_equiv_gaussian_sigma(logp);
    } else {
        int which, status;
        double p, q, bound, mean = 0.0, sd = 1.0;
        q = exp(logp);
        p = 1.0 - q;
        which = 2;
        status = 0;
        /* Convert to a sigma */
        cdfnor(&which, &p, &q, &x, &mean, &sd, &status, &bound);
        if (status) {
            if (status == -2) {
                x = 0.0;
            } else if (status == -3) {
                x = 38.5;
            } else {
                printf("\nError in cdfnor() (candidate_sigma()):\n");
                printf("   status = %d, bound = %g\n", status, bound);
                printf("   p = %g, q = %g, x = %g, mean = %g, sd = %g\n\n",
                       p, q, x, mean, sd);
                exit(1);
            }
        }
    }
    if (x < 0.0)
        return 0.0;
    else
        return x;
}

double __device__ power_to_logp(float chi2, float dof){
    double double_dof = (double) dof;
    double double_chi2 = (double) chi2;
    // Use boundary condition
    if (dof >= chi2 * 1.05){
        return 0.0;
    } else {
        double x = 1500 * double_dof / double_chi2;
        // Updated polynomial equation
        double f_x = (-4.460405902717228e-46 * pow(x, 16) + 9.492786384945832e-42 * pow(x, 15) - 
               9.147045144529116e-38 * pow(x, 14) + 5.281085384219971e-34 * pow(x, 13) - 
               2.0376166670276118e-30 * pow(x, 12) + 5.548033164083744e-27 * pow(x, 11) - 
               1.0973877021703706e-23 * pow(x, 10) + 1.5991806841151474e-20 * pow(x, 9) - 
               1.7231488066853853e-17 * pow(x, 8) + 1.3660070957914896e-14 * pow(x, 7) - 
               7.861795249869729e-12 * pow(x, 6) + 3.2136336591718867e-09 * pow(x, 5) - 
               9.046641813341226e-07 * pow(x, 4) + 0.00016945948004599545 * pow(x, 3) - 
               0.0214942314851717 * pow(x, 2) + 2.951595476316614 * x - 
               755.240918031251);
        double logp = chi2 * f_x / 1500;
        return logp;
    }
}

__global__ void wakeGPUKernel(){
    // This kernel does nothing, it is used to wake up the GPU
    // so that the first kernel run is not slow
}

__global__ void separateRealAndImaginaryComponents(half2* rawDataDevice, half* realData, half* imaginaryData, long numComplexFloats){
    long globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;
    if (globalThreadIndex < numComplexFloats){
        half2 currentValue = rawDataDevice[globalThreadIndex];
        realData[globalThreadIndex] = currentValue.x;
        imaginaryData[globalThreadIndex] = currentValue.y;

        // check for inf or nan
        /*if (isinf((float)realData[globalThreadIndex]) || isnan((float)realData[globalThreadIndex])){
            printf("realData1[%ld] = %f\n", globalThreadIndex, (float)realData[globalThreadIndex]);
        }
        if (isinf((float)imaginaryData[globalThreadIndex]) || isnan((float)imaginaryData[globalThreadIndex])){
            printf("imaginaryData1[%ld] = %f\n", globalThreadIndex, (float)imaginaryData[globalThreadIndex]);
        }*/
    }
}



__global__ void medianOfMediansNormalisation(half* globalArray) {
    // HARDCODED FOR PERFORMANCE
    // USE medianOfMediansNormalisationAnyBlockSize() FOR GENERAL USE

    // Each thread loads 4 elements from global memory to shared memory
    // then calculates the median of these 4 elements, recursively reducing the array down to 
    //      a single median of medians value
    // then subtracts the median of medians from each element
    // then takes the absolute value of each element
    // then calculates the median of these absolute values
    // then multiplies this new median (aka median absolute deviation) by 1.4826
    // then subtracts the median from each original element and divides by the new median absolute deviation

    // Assumes blockDim.x = 1024
    // TODO: make this work for any blockDim.x
    __shared__ half medianArray[4096];
    __shared__ half madArray[4096];
    __shared__ half normalisedArray[4096];

    //int globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;
    int localThreadIndex = threadIdx.x;
    int globalArrayIndex = blockDim.x*blockIdx.x*4+threadIdx.x;

    half median;
    half mad;

    medianArray[localThreadIndex] = globalArray[globalArrayIndex];
    medianArray[localThreadIndex + 1024] = globalArray[globalArrayIndex + 1024];
    medianArray[localThreadIndex + 2048] = globalArray[globalArrayIndex + 2048];
    medianArray[localThreadIndex + 3072] = globalArray[globalArrayIndex + 3072];

    madArray[localThreadIndex] = medianArray[localThreadIndex];
    madArray[localThreadIndex + 1024] = medianArray[localThreadIndex + 1024];
    madArray[localThreadIndex + 2048] = medianArray[localThreadIndex + 2048];
    madArray[localThreadIndex + 3072] = medianArray[localThreadIndex + 3072];

    normalisedArray[localThreadIndex] = medianArray[localThreadIndex];
    normalisedArray[localThreadIndex + 1024] = medianArray[localThreadIndex + 1024];
    normalisedArray[localThreadIndex + 2048] = medianArray[localThreadIndex + 2048];
    normalisedArray[localThreadIndex + 3072] = medianArray[localThreadIndex + 3072];

    __syncthreads();

    half a,b,c,d,min,max;
  

    for (int upperThreadIndex = 1024; upperThreadIndex > 0; upperThreadIndex >>=2){
        if(localThreadIndex < upperThreadIndex){
            a = medianArray[localThreadIndex];
            b = medianArray[localThreadIndex+upperThreadIndex];
            c = medianArray[localThreadIndex+upperThreadIndex*2];
            d = medianArray[localThreadIndex+upperThreadIndex*3];
            min = __hmin(__hmin(__hmin(a,b),c),d);
            max = __hmax(__hmax(__hmax(a,b),c),d);
            medianArray[localThreadIndex] = (a+b+c+d-min-max)*((half)0.5);
        }
        __syncthreads();
    }


    median = medianArray[0];
    __syncthreads();

    madArray[localThreadIndex] = __habs(madArray[localThreadIndex] - median);
    madArray[localThreadIndex + 1024] = __habs(madArray[localThreadIndex + 1024] - median);
    madArray[localThreadIndex + 2048] = __habs(madArray[localThreadIndex + 2048] - median);
    madArray[localThreadIndex + 3072] = __habs(madArray[localThreadIndex + 3072] - median);

    __syncthreads();
    
    for (int upperThreadIndex = 1024; upperThreadIndex > 0; upperThreadIndex >>=2){
        if(localThreadIndex < upperThreadIndex){
            a = madArray[localThreadIndex];
            b = madArray[localThreadIndex+upperThreadIndex];
            c = madArray[localThreadIndex+upperThreadIndex*2];
            d = madArray[localThreadIndex+upperThreadIndex*3];
            min = __hmin(__hmin(__hmin(a,b),c),d);
            max = __hmax(__hmax(__hmax(a,b),c),d);
            madArray[localThreadIndex] = (a+b+c+d-min-max)*((half)0.5);
        }
        __syncthreads();
    }
    
    mad =  madArray[0]*((half)1.4826);
    __syncthreads();


    normalisedArray[localThreadIndex] = (normalisedArray[localThreadIndex] - median) / mad;
    normalisedArray[localThreadIndex + 1024] = (normalisedArray[localThreadIndex + 1024] - median) / mad;
    normalisedArray[localThreadIndex + 2048] = (normalisedArray[localThreadIndex + 2048] - median) / mad;
    normalisedArray[localThreadIndex + 3072] = (normalisedArray[localThreadIndex + 3072] - median) / mad;

    __syncthreads();

    globalArray[globalArrayIndex] = normalisedArray[localThreadIndex];
    globalArray[globalArrayIndex + 1024] = normalisedArray[localThreadIndex + 1024];
    globalArray[globalArrayIndex + 2048] = normalisedArray[localThreadIndex + 2048];
    globalArray[globalArrayIndex + 3072] = normalisedArray[localThreadIndex + 3072];

}


__global__ void magnitudeSquared(half* realData, half* imaginaryData, half* magnitudeSquaredArray, long numFloats){
    int globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;
    if (globalThreadIndex < numFloats){
        half real = realData[globalThreadIndex];
        half imaginary = imaginaryData[globalThreadIndex];
        magnitudeSquaredArray[globalThreadIndex] = real*real + imaginary*imaginary;


        // check for inf or nan
        /*if (isinf((float)magnitudeSquaredArray[globalThreadIndex]) || isnan((float)magnitudeSquaredArray[globalThreadIndex])){
                printf("magnitudeSquaredArray[%ld] = %f\n", globalThreadIndex, (float)magnitudeSquaredArray[globalThreadIndex]);
                printf("realData[%ld] = %f\n", globalThreadIndex, (float)realData[globalThreadIndex]);
                printf("imaginaryData[%ld] = %f\n", globalThreadIndex, (float)imaginaryData[globalThreadIndex]);
        }*/
    }
}


// takes a 1D array like this:
// magnitudeSquaredArray:   [0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0]
// and adds these elements together, effectively performing a harmonic sum
// decimatedArray2:         [0,0,0,0,0,x,0,0,0,0,0,x,x,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0]
// decimatedArray3:         [0,0,0,0,0,x,0,0,0,0,0,x,x,0,0,0,0,x,x,x,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0]
// decimatedArray4:         [0,0,0,0,0,x,0,0,0,0,0,x,x,0,0,0,0,x,x,x,0,0,0,x,x,x,x,0,0,0,0,0,0,0,0,0]
//                                     |<--------->|<--------->|<--------->|
//                                        equal spacing between harmonics

__global__ void decimateHarmonics(half* magnitudeSquaredArray, half* decimatedArray2, half* decimatedArray3, half* decimatedArray4, long numMagnitudes){
    int globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;

    half fundamental;
    half harmonic1a, harmonic1b;
    half harmonic2a, harmonic2b, harmonic2c;
    half harmonic3a, harmonic3b, harmonic3c, harmonic3d;

    if (globalThreadIndex*2+1 < numMagnitudes){
        fundamental = magnitudeSquaredArray[globalThreadIndex];
        harmonic1a = magnitudeSquaredArray[globalThreadIndex*2];
        harmonic1b = magnitudeSquaredArray[globalThreadIndex*2+1];
        decimatedArray2[globalThreadIndex] = fundamental+harmonic1a+harmonic1b;
    }

    if (globalThreadIndex*3+2 < numMagnitudes){
        harmonic2a = magnitudeSquaredArray[globalThreadIndex*3];
        harmonic2b = magnitudeSquaredArray[globalThreadIndex*3+1];
        harmonic2c = magnitudeSquaredArray[globalThreadIndex*3+2];
        decimatedArray3[globalThreadIndex] = fundamental+harmonic1a+harmonic1b
                                                +harmonic2a+harmonic2b+harmonic2c;
    }

    if (globalThreadIndex*4+3 < numMagnitudes){
        harmonic3a = magnitudeSquaredArray[globalThreadIndex*4];
        harmonic3b = magnitudeSquaredArray[globalThreadIndex*4+1];
        harmonic3c = magnitudeSquaredArray[globalThreadIndex*4+2];
        harmonic3d = magnitudeSquaredArray[globalThreadIndex*4+3];
        decimatedArray4[globalThreadIndex] = fundamental+harmonic1a+harmonic1b
                                                +harmonic2a+harmonic2b+harmonic2c
                                                +harmonic3a+harmonic3b+harmonic3c+harmonic3d;
    }

    // check if any of the inputs are inf or nan
    /*if (isinf((float)fundamental) || isnan((float)fundamental)){
        printf("fundamental[%d] = %f\n", globalThreadIndex, (float)fundamental);
    }*/

}

// I WANT TO __FORCEINLINE__ THIS FUNCTION BUT APPARENTLY YOU CAN'T INLINE FUNCTIONS WITH A __SYNCTHREADS() IN
__device__ void searchAndUpdate(half* sumArray, power_index_struct* searchArray, candidate* localCandidateArray, int z, int outputCounter, int localThreadIndex, int globalThreadIndex, int numharm){
    searchArray[localThreadIndex].power = sumArray[localThreadIndex];
    searchArray[localThreadIndex].index = localThreadIndex;
    for (int stride = blockDim.x / 2; stride>0; stride /= 2){
        if (localThreadIndex < stride){
            if (searchArray[localThreadIndex].power < searchArray[localThreadIndex + stride].power){
                searchArray[localThreadIndex] = searchArray[localThreadIndex + stride];
            }
        }
        __syncthreads();
    }
    if (localThreadIndex == 0){
        localCandidateArray[outputCounter].power = (half)(searchArray[0].power);
        localCandidateArray[outputCounter].r = blockIdx.x * blockDim.x + (int) searchArray[0].index;
        localCandidateArray[outputCounter].z = z;
        localCandidateArray[outputCounter].logp = 0.0f;
        localCandidateArray[outputCounter].numharm = numharm;
    }
}


__global__ void boxcarFilterArray(half* magnitudeSquaredArray, candidate* globalCandidateArray, int numharm, long numFloats, int numCandidatesPerBlock){
    __shared__ half lookupArray[512];
    __shared__ half sumArray[256];
    __shared__ power_index_struct searchArray[256];
    __shared__ candidate localCandidateArray[16]; //oversized, has to be greater than numCandidatesPerBlock

    int globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;
    int localThreadIndex = threadIdx.x;

    lookupArray[localThreadIndex] = magnitudeSquaredArray[globalThreadIndex];
    lookupArray[localThreadIndex + 256] = magnitudeSquaredArray[globalThreadIndex + 256];
    if (globalThreadIndex < numFloats) {
        lookupArray[threadIdx.x] = magnitudeSquaredArray[globalThreadIndex];
        // check for inf or nan
        /*if (isinf( (float)magnitudeSquaredArray[globalThreadIndex]) || isnan( (float)magnitudeSquaredArray[globalThreadIndex])){
            printf("magnitudeSquaredArray[%d] = %f\n", globalThreadIndex, (float)magnitudeSquaredArray[globalThreadIndex]);
        }*/
    } else {
        lookupArray[threadIdx.x] = 0.0f;
    }

    if (globalThreadIndex + 256 < numFloats) {
        lookupArray[threadIdx.x + 256] = magnitudeSquaredArray[globalThreadIndex + 256];
    } else {
        lookupArray[threadIdx.x + 256] = 0.0f;
    }
    

    __syncthreads();

    // initialise the sum array
    sumArray[localThreadIndex] = 0.0f;
    __syncthreads();

    // begin boxcar filtering
    // search at z = 0
    sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + 0];
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 0, 0, localThreadIndex, globalThreadIndex, numharm);

    // search at z = 1
    sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + 1];
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 1, 1, localThreadIndex, globalThreadIndex, numharm);

    // search at z = 2
    sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + 2];
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 2, 2, localThreadIndex, globalThreadIndex, numharm);

    // search at z = 4
    sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + 3];
    sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + 4];
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 4, 3, localThreadIndex, globalThreadIndex, numharm);

    // search at z = 8
    sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + 5];
    sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + 6];
    sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + 7];
    sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + 8];
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 8, 4, localThreadIndex, globalThreadIndex, numharm);

    // search at z = 16
    #pragma unroll
    for (int z = 9; z < 17; z++){
        sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + z];
    }
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 16, 5, localThreadIndex, globalThreadIndex, numharm);

    // search at z = 32
    #pragma unroll
    for (int z = 17; z < 33; z++){
        sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + z];
    }
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 32, 6, localThreadIndex, globalThreadIndex, numharm);

    // search at z = 64
    #pragma unroll
    for (int z = 33; z < 65; z++){
        sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + z];
    }
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 64, 7, localThreadIndex, globalThreadIndex, numharm);

    // search at z = 128
    #pragma unroll
    for (int z = 65; z < 129; z++){
        sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + z];
    }
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 128, 8, localThreadIndex, globalThreadIndex, numharm);

    // search at z = 256
    #pragma unroll
    for (int z = 129; z < 257; z++){
        sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + z];
    }
    __syncthreads();
    searchAndUpdate(sumArray, searchArray, localCandidateArray, 256, 9, localThreadIndex, globalThreadIndex, numharm);

    __syncthreads();

    if (localThreadIndex < numCandidatesPerBlock){
        globalCandidateArray[blockIdx.x*numCandidatesPerBlock+localThreadIndex] = localCandidateArray[localThreadIndex];
    }
}


__global__ void calculateLogp(candidate* globalCandidateArray, long numCandidates, int numSum){
    int globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;
    if (globalThreadIndex < numCandidates){
        double logp = power_to_logp(globalCandidateArray[globalThreadIndex].power,globalCandidateArray[globalThreadIndex].z*numSum*2);
        globalCandidateArray[globalThreadIndex].logp = (float) logp;
    }
}

__global__ void convertFP32ArrayToFP16Array(float* inputArray, half* outputArray, long numFloats){
    int globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;
    if (globalThreadIndex < numFloats){
        outputArray[globalThreadIndex] = __float2half(inputArray[globalThreadIndex]);
        // check for inf or nan
        if (isinf(inputArray[globalThreadIndex]) || isnan(inputArray[globalThreadIndex])){
            printf("inputArray3[%d] = %f\n", globalThreadIndex, inputArray[globalThreadIndex]);
            printf("outputArray3[%d] = %f\n", globalThreadIndex, __half2float(outputArray[globalThreadIndex]));
        }
        if (isinf(__half2float(outputArray[globalThreadIndex])) || isnan(__half2float(outputArray[globalThreadIndex]))){
            printf("inputArray4[%d] = %f\n", globalThreadIndex, inputArray[globalThreadIndex]);
            printf("outputArray4[%d] = %f\n", globalThreadIndex, __half2float(outputArray[globalThreadIndex]));
        }
    }
}

void copyDeviceArrayToHostAndPrint(float* deviceArray, long numFloats){
    float* hostArray;
    hostArray = (float*)malloc(sizeof(float)*numFloats);
    hipMemcpy(hostArray, deviceArray, sizeof(float)*numFloats,hipMemcpyDeviceToHost);
    for (int i = 0; i < numFloats; i++){
        printf("%f\n", hostArray[i]);
    }
    free(hostArray);
}

void copyDeviceArrayToHostAndSaveToFile(float* deviceArray, long numFloats, const char* filename){
    float* hostArray;
    hostArray = (float*)malloc(sizeof(float)*numFloats);
    hipMemcpy(hostArray, deviceArray, sizeof(float)*numFloats,hipMemcpyDeviceToHost);
    FILE *f = fopen(filename, "wb");
    // write in csv format, one number per column
    for (int i = 0; i < numFloats; i++){
        fprintf(f, "%f\n", hostArray[i]);
    }
    fclose(f);
    free(hostArray);
}

void copyDeviceCandidateArrayToHostAndPrint(candidate* deviceArray, long numCandidates){
    candidate* hostArray;
    hostArray = (candidate*)malloc(sizeof(candidate)*numCandidates);
    hipMemcpy(hostArray, deviceArray, sizeof(candidate)*numCandidates,hipMemcpyDeviceToHost);
    for (int i = 0; i < numCandidates; i++){
        printf("Candidate %d: power: %f, logp: %f, r: %d, z: %d, numharm: %d\n", i, hostArray[i].power, hostArray[i].logp, hostArray[i].r, hostArray[i].z, hostArray[i].numharm);
    }
    free(hostArray);
}

#define RESET   "\033[0m"
#define FLASHING   "\033[5m"
#define BOLD   "\033[1m"

// (Discard DC kernel definition)
__global__ void discardDC_kernel(const float* __restrict__ d_in,
                                 float* __restrict__ d_out,
                                 int outCount)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < outCount-2) {
        d_out[idx] = d_in[idx + 2]; 
    }
}


long readDatAndDiscardDC(const char* filepath, float** d_out)
{
    FILE *f = fopen(filepath, "rb");
    if (!f) {
        fprintf(stderr, "Failed to open .dat file: %s\n", filepath);
        return -1;
    }
    fseek(f, 0, SEEK_END);
    long fileSizeBytes = ftell(f);
    fseek(f, 0, SEEK_SET);

    long N = fileSizeBytes / sizeof(float);
    if (N < 2) {
        fprintf(stderr, "Not enough data in .dat file.\n");
        fclose(f);
        return -1;
    }

    // Optionally align down to multiple of 8192 if desired:
    // N -= (N % 8192);

    float* h_timeData = (float*)malloc(N*sizeof(float));
    size_t itemsRead = fread(h_timeData, sizeof(float), N, f);
    fclose(f);
    if (itemsRead != (size_t)N) {
        fprintf(stderr, "Error reading .dat: only %zu of %ld floats read\n",
                itemsRead, N);
        free(h_timeData);
        return -1;
    }

    printf("N+2 = %ld\n", N+2);

    float* d_timeData = NULL;
    hipMalloc((void**)&d_timeData, N*sizeof(float));
    hipMemcpy(d_timeData, h_timeData, N*sizeof(float), hipMemcpyHostToDevice);
    free(h_timeData);

    float* d_freqTemp = NULL; 
    hipMalloc((void**)&d_freqTemp, (N+2)*sizeof(float)); // R2C => N+2 floats

    hipfftHandle plan;
    hipfftPlan1d(&plan, (int)N, HIPFFT_R2C, 1);
    hipfftResult res = hipfftExecR2C(plan, (hipfftReal*)d_timeData, (hipfftComplex*)d_freqTemp);
    hipfftDestroy(plan);
    hipFree(d_timeData);
    if (res != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftExecR2C failed.\n");
        hipFree(d_freqTemp);
        return -1;
    }


    // Allocate final array for "no-DC" => length N floats
    float* d_freqNoDC = NULL;
    hipMalloc((void**)&d_freqNoDC, N*sizeof(float));

    int blockSize = 256;
    int gridSize = (int)((N + blockSize - 1)/blockSize);
    discardDC_kernel<<<gridSize, blockSize>>>(d_freqTemp, d_freqNoDC, (int)N);

    hipFree(d_freqTemp);

    *d_out = d_freqNoDC;



    return N; // the final array has N floats (i.e. (N/2) complex bins)
}


// Pulscan ASCII
const char* pulscan_frame = 
"    .          .     .     *        .   .   .     .\n"
"         " BOLD "___________      . __" RESET " .  .   *  .   .  .  .     .\n"
"    . *   " BOLD "_____  __ \\__+ __/ /_____________ _____" RESET " .    " FLASHING "*" RESET "  .\n"
"  +    .   " BOLD "___  /_/ / / / / / ___/ ___/ __ `/ __ \\" RESET "     + .\n"
" .          " BOLD "_  ____/ /_/ / (__  ) /__/ /_/ / / / /" RESET " .  *     . \n"
"       .    " BOLD "/_/ *  \\__,_/_/____/\\___/\\__,_/_/ /_/" RESET "    \n"
"    *    +     .     .     . +     .     +   .      *   +\n"
"  J. White, K. Adámek, J. Roy, S. Ransom, W. Armour  2023\n\n";

int main(int argc, char* argv[]){
    printf("%s\n", pulscan_frame);

    // start high resolution timer to measure gpu initialisation time using chrono
    auto start_chrono = std::chrono::high_resolution_clock::now();
    
    hipDeviceSynchronize();
    wakeGPUKernel<<<1,1>>>();

    auto end_chrono = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono);
    
    printf("GPU initialisation took:                %f ms\n",(float)duration.count());
    
    // start timing
    start_chrono = std::chrono::high_resolution_clock::now();

    if (argc < 2) {
        printf("Please provide the input file path as a command line argument.\n");
        return 1;
    }
    const char* filepath = argv[1];
    
    // Detect file extension
    size_t filepathLen = strlen(filepath);
    bool isFFT = false, isDAT = false;
    if (filepathLen > 4) {
        const char* ext = filepath + (filepathLen - 4);
        if (strcmp(ext, ".fft") == 0) {
            isFFT = true;
        } else if (strcmp(ext, ".dat") == 0) {
            isDAT = true;
        }
    }
    if (!isFFT && !isDAT) {
        printf("Error: input file must end with .fft or .dat\n");
        return 1;
    }

    float* rawDataDevice = NULL;
    size_t numFloats = 0; // total floats in rawDataDevice

    if (isFFT) {
        FILE *f = fopen(filepath, "rb");
        if (!f) {
            fprintf(stderr, "Failed to open .fft file: %s\n", filepath);
            return 1;
        }

        fseek(f, 0, SEEK_END);
        size_t filesize = ftell(f);
        fseek(f, 0, SEEK_SET);
        
        numFloats = filesize / sizeof(float);

        // Cap the filesize at the nearest lower factor of 8192
        numFloats = numFloats - (numFloats % 8192);

        float* rawDataHost = (float*) malloc(sizeof(float)*numFloats);
        size_t itemsRead = fread(rawDataHost, sizeof(float), numFloats, f);
        fclose(f);
        if (itemsRead != numFloats) {
            fprintf(stderr, "Error reading .fft file: only %zu of %zu items read\n",
                    itemsRead, numFloats);
        }
        end_chrono = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono);
        printf("Reading file took:                      %f ms\n", (float)duration.count());

        // Copy raw .fft data to GPU
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        hipMalloc((void**)&rawDataDevice, sizeof(float)*numFloats);
        hipMemcpy(rawDataDevice, rawDataHost, sizeof(float)*numFloats, hipMemcpyHostToDevice);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Copying data to GPU took:               %f ms\n", milliseconds);

        free(rawDataHost);

    } else {

        // read the .dat file, run R2C, discard DC => get an array of length N floats
        // This also includes the time to copy to GPU, so let's time it
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        auto start_chrono = std::chrono::high_resolution_clock::now();

        long N = readDatAndDiscardDC(filepath, &rawDataDevice);

        auto end_chrono = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono);
        printf("Reading file took:                      %f ms\n", (float)duration.count());

        if (N <= 0) {
            fprintf(stderr, "readDatAndDiscardDC failed.\n");
            return 1;
        }
        numFloats = (size_t)N;  // final # of floats in rawDataDevice

        // Cap the floats at the nearest lower factor of 8192
        numFloats = numFloats - (numFloats % 8192);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Performing R2C & discarding DC took:    %f ms\n", milliseconds);
    }

    half* rawDataDevice_fp16;
    hipMalloc((void**)&rawDataDevice_fp16, sizeof(half)*numFloats);

    // Now we have:
    //   rawDataDevice: interleaved complex floats
    //   numFloats: total floats in that array
    //
    // The remainder of the pipeline is unchanged.

    // Start measuring GPU pipeline
    hipEvent_t start, stop, overallGPUStart, overallGPUStop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    

    hipEventCreate(&overallGPUStart);
    hipEventCreate(&overallGPUStop);
    hipEventRecord(overallGPUStart);

    // convert rawDataDevice to half
    int numThreadsConvert = 256;
    int numBlocksConvert = (numFloats + numThreadsConvert - 1)/ numThreadsConvert;

    convertFP32ArrayToFP16Array<<<numBlocksConvert, numThreadsConvert>>>(rawDataDevice, rawDataDevice_fp16, numFloats);

    int numMagnitudes = numFloats/2;  // each complex bin has 2 floats
    printf("Number of magnitude bins:               %d\n", numMagnitudes);

    // 1) Separate real & imaginary
    half* realDataDevice;
    half* imaginaryDataDevice;
    hipMalloc((void**)&realDataDevice, sizeof(half)*numMagnitudes);
    hipMalloc((void**)&imaginaryDataDevice, sizeof(half)*numMagnitudes);

    int numThreadsSeparate = 256;
    int numBlocksSeparate = (numMagnitudes + numThreadsSeparate - 1)/ numThreadsSeparate;
    hipEventRecord(start);
    separateRealAndImaginaryComponents<<<numBlocksSeparate, numThreadsSeparate>>>(
        (half2*)rawDataDevice_fp16, realDataDevice, imaginaryDataDevice, numMagnitudes);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Separating complex components took:     %f ms\n", milliseconds);

    // 2) Normalise real and imaginary parts
    
    int numThreadsNormalise = 1024; // must be 1024 for kernel
    int numMagnitudesPerThreadNormalise = 4;
    int numBlocksNormalise = ((numMagnitudes/numMagnitudesPerThreadNormalise)
                              + numThreadsNormalise - 1)/ numThreadsNormalise;

    hipEventRecord(start);
    medianOfMediansNormalisation<<<numBlocksNormalise, numThreadsNormalise>>>(realDataDevice);
    medianOfMediansNormalisation<<<numBlocksNormalise, numThreadsNormalise>>>(imaginaryDataDevice);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Normalisation took:                     %f ms\n", milliseconds);

    // 3) Magnitude
    
    half* magnitudeSquaredArray;
    hipMalloc((void**)&magnitudeSquaredArray, sizeof(half)*numMagnitudes);

    int numThreadsMagnitude = 1024;
    int numBlocksMagnitude = (numMagnitudes + numThreadsMagnitude - 1)/ numThreadsMagnitude;

    hipEventRecord(start);
    magnitudeSquared<<<numBlocksMagnitude, numThreadsMagnitude>>>(
        realDataDevice, imaginaryDataDevice, magnitudeSquaredArray, numMagnitudes);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Magnitude took:                         %f ms\n", milliseconds);

    // 4) Decimate for harmonic summation
    
    half* decimatedArrayBy2;
    half* decimatedArrayBy3;
    half* decimatedArrayBy4;
    hipMalloc((void**)&decimatedArrayBy2, sizeof(half)* (numMagnitudes/2));
    hipMalloc((void**)&decimatedArrayBy3, sizeof(half)* (numMagnitudes/3));
    hipMalloc((void**)&decimatedArrayBy4, sizeof(half)* (numMagnitudes/4));

    int numThreadsDecimate = 256;
    int numBlocksDecimate = (numMagnitudes/2 + numThreadsDecimate - 1)/ numThreadsDecimate;
    hipEventRecord(start);
    decimateHarmonics<<<numBlocksDecimate, numThreadsDecimate>>>(
        magnitudeSquaredArray, decimatedArrayBy2, decimatedArrayBy3, decimatedArrayBy4, 
        numMagnitudes);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Decimation took:                        %f ms\n", milliseconds);

    long numFloats2 = numMagnitudes/2;
    numFloats2 = numFloats2 - (numFloats2 % 8192);

    long numFloats3 = numMagnitudes/3;
    numFloats3 = numFloats3 - (numFloats3 % 8192);

    long numFloats4 = numMagnitudes/4;
    numFloats4 = numFloats4 - (numFloats4 % 8192);


    // 5) Boxcar filtering on each harmonic array (1,2,3,4)

    int numThreadsBoxcar = 256;
    int numBlocksBoxcar1 = (numMagnitudes + numThreadsBoxcar - 1)/ numThreadsBoxcar;
    int numBlocksBoxcar2 = ((numFloats2) + numThreadsBoxcar - 1)/ numThreadsBoxcar;
    int numBlocksBoxcar3 = ((numFloats3) + numThreadsBoxcar - 1)/ numThreadsBoxcar;
    int numBlocksBoxcar4 = ((numFloats4) + numThreadsBoxcar - 1)/ numThreadsBoxcar;

    candidate* globalCandidateArray1;
    candidate* globalCandidateArray2;
    candidate* globalCandidateArray3;
    candidate* globalCandidateArray4;

    int zmax = 256;
    int numCandidatesPerBlock = 1;
    int i = 1;
    while (i <= zmax){
        i *= 2;
        numCandidatesPerBlock++;
    }

    hipMalloc((void**)&globalCandidateArray1, sizeof(candidate)*numCandidatesPerBlock*numBlocksBoxcar1);
    hipMalloc((void**)&globalCandidateArray2, sizeof(candidate)*numCandidatesPerBlock*numBlocksBoxcar2);
    hipMalloc((void**)&globalCandidateArray3, sizeof(candidate)*numCandidatesPerBlock*numBlocksBoxcar3);
    hipMalloc((void**)&globalCandidateArray4, sizeof(candidate)*numCandidatesPerBlock*numBlocksBoxcar4);

    hipMemset(globalCandidateArray1, 0, sizeof(candidate)*numCandidatesPerBlock*numBlocksBoxcar1);
    hipMemset(globalCandidateArray2, 0, sizeof(candidate)*numCandidatesPerBlock*numBlocksBoxcar2);
    hipMemset(globalCandidateArray3, 0, sizeof(candidate)*numCandidatesPerBlock*numBlocksBoxcar3);
    hipMemset(globalCandidateArray4, 0, sizeof(candidate)*numCandidatesPerBlock*numBlocksBoxcar4);
    hipEventRecord(start);

    // Launch boxcar for each harmonic array
    boxcarFilterArray<<<numBlocksBoxcar1, numThreadsBoxcar, 0>>>(
        magnitudeSquaredArray, globalCandidateArray1, 1, numMagnitudes, numCandidatesPerBlock);
    boxcarFilterArray<<<numBlocksBoxcar2, numThreadsBoxcar, 0>>>(
        decimatedArrayBy2, globalCandidateArray2, 2, numMagnitudes/2, numCandidatesPerBlock);
    boxcarFilterArray<<<numBlocksBoxcar3, numThreadsBoxcar, 0>>>(
        decimatedArrayBy3, globalCandidateArray3, 3, numMagnitudes/3, numCandidatesPerBlock);
    boxcarFilterArray<<<numBlocksBoxcar4, numThreadsBoxcar, 0>>>(
        decimatedArrayBy4, globalCandidateArray4, 4, numMagnitudes/4, numCandidatesPerBlock);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Boxcar filtering took:                  %f ms\n", milliseconds);

    // 6) Calculate logp for each candidate
    
    int numThreadsLogp = 256;
    int totalCands1 = numBlocksBoxcar1 * numCandidatesPerBlock;
    int totalCands2 = numBlocksBoxcar2 * numCandidatesPerBlock;
    int totalCands3 = numBlocksBoxcar3 * numCandidatesPerBlock;
    int totalCands4 = numBlocksBoxcar4 * numCandidatesPerBlock;

    int numBlocksLogp1 = (totalCands1 + numThreadsLogp - 1)/ numThreadsLogp;
    int numBlocksLogp2 = (totalCands2 + numThreadsLogp - 1)/ numThreadsLogp;
    int numBlocksLogp3 = (totalCands3 + numThreadsLogp - 1)/ numThreadsLogp;
    int numBlocksLogp4 = (totalCands4 + numThreadsLogp - 1)/ numThreadsLogp;

    hipEventRecord(start);
    calculateLogp<<<numBlocksLogp1, numThreadsLogp, 0>>>(
        globalCandidateArray1, totalCands1, 1);
    calculateLogp<<<numBlocksLogp2, numThreadsLogp, 0>>>(
        globalCandidateArray2, totalCands2, 3);
    calculateLogp<<<numBlocksLogp3, numThreadsLogp, 0>>>(
        globalCandidateArray3, totalCands3, 6);
    calculateLogp<<<numBlocksLogp4, numThreadsLogp, 0>>>(
        globalCandidateArray4, totalCands4, 10);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Logp time taken:                        %f ms\n", milliseconds);

    // stop overall GPU timing
    hipEventRecord(overallGPUStop);
    hipEventSynchronize(overallGPUStop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, overallGPUStart, overallGPUStop);
    printf("Overall GPU time taken:                 %f ms\n", milliseconds);

    // ----------------------------------------------------
    // Write output file
    // ----------------------------------------------------
    auto write_start = std::chrono::high_resolution_clock::now();

    candidate* hostCandidateArray1 = (candidate*)malloc(sizeof(candidate)*totalCands1);
    candidate* hostCandidateArray2 = (candidate*)malloc(sizeof(candidate)*totalCands2);
    candidate* hostCandidateArray3 = (candidate*)malloc(sizeof(candidate)*totalCands3);
    candidate* hostCandidateArray4 = (candidate*)malloc(sizeof(candidate)*totalCands4);

    hipMemcpy(hostCandidateArray1, globalCandidateArray1, sizeof(candidate)*totalCands1, hipMemcpyDeviceToHost);
    hipMemcpy(hostCandidateArray2, globalCandidateArray2, sizeof(candidate)*totalCands2, hipMemcpyDeviceToHost);
    hipMemcpy(hostCandidateArray3, globalCandidateArray3, sizeof(candidate)*totalCands3, hipMemcpyDeviceToHost);
    hipMemcpy(hostCandidateArray4, globalCandidateArray4, sizeof(candidate)*totalCands4, hipMemcpyDeviceToHost);

    // Build output filename
    char outputFilename[256];
    snprintf(outputFilename, sizeof(outputFilename), "%s", filepath);
    if (filepathLen > 4) {
        // remove the .fft or .dat suffix
        outputFilename[filepathLen - 4] = '\0';
    }
    strncat(outputFilename, ".gpucand", sizeof(outputFilename) - strlen(outputFilename) - 1);

    FILE *csvFile = fopen(outputFilename, "w");
    fprintf(csvFile, "sigma,logp,r,z,power,numharm\n");

    float logpThreshold = -10;

    // Gather final candidates
    int totalSize = totalCands1 + totalCands2 + totalCands3 + totalCands4;
    candidate* finalCandidateArray = (candidate*)malloc(sizeof(candidate)*totalSize);

    int candidateCounter = 0;
    #define ADD_CANDIDATES_FROM(Array, Count) \
        for (int i = 0; i < (Count); i++){ \
            if ((Array)[i].logp < logpThreshold && (Array)[i].r != 0 && (Array)[i].z != 0) { \
                finalCandidateArray[candidateCounter++] = (Array)[i]; \
            } \
        }

    ADD_CANDIDATES_FROM(hostCandidateArray1, totalCands1);
    ADD_CANDIDATES_FROM(hostCandidateArray2, totalCands2);
    ADD_CANDIDATES_FROM(hostCandidateArray3, totalCands3);
    ADD_CANDIDATES_FROM(hostCandidateArray4, totalCands4);

    // sort by logp ascending
    qsort(finalCandidateArray, candidateCounter, sizeof(candidate), compareCandidatesByLogp);

    // write out
    for (int i = 0; i < candidateCounter; i++){
        double sig = equivalent_gaussian_sigma((double) finalCandidateArray[i].logp);
        fprintf(csvFile, "%lf,%f,%d,%d,%f,%d\n",
                sig, finalCandidateArray[i].logp,
                finalCandidateArray[i].r, finalCandidateArray[i].z,
                finalCandidateArray[i].power, finalCandidateArray[i].numharm);
    }
    fclose(csvFile);

    auto write_end = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::milliseconds>(write_end - write_start);
    printf("Writing output file took:               %f ms\n", (float)duration.count());

    // Cleanup
    free(hostCandidateArray1);
    free(hostCandidateArray2);
    free(hostCandidateArray3);
    free(hostCandidateArray4);
    free(finalCandidateArray);

    hipFree(rawDataDevice);
    hipFree(realDataDevice);
    hipFree(imaginaryDataDevice);
    hipFree(magnitudeSquaredArray);
    hipFree(decimatedArrayBy2);
    hipFree(decimatedArrayBy3);
    hipFree(decimatedArrayBy4);
    hipFree(globalCandidateArray1);
    hipFree(globalCandidateArray2);
    hipFree(globalCandidateArray3);
    hipFree(globalCandidateArray4);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    return 0;
}
